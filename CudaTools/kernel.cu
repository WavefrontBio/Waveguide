#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdint.h>

#include <iostream>
#include <hip/hip_runtime.h>
#include <>
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>


/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Utility Functions
struct GpuTimer
{
	hipEvent_t start;
	hipEvent_t stop;

	GpuTimer()
	{
		hipEventCreate(&start);
		hipEventCreate(&stop);
	}

	~GpuTimer()
	{
		hipEventDestroy(start);
		hipEventDestroy(stop);
	}

	void Start()
	{
		hipEventRecord(start, 0);
	}

	void Stop()
	{
		hipEventRecord(stop, 0);
	}

	float ElapsedMillis()
	{
		float elapsed;
		hipEventSynchronize(stop);
		hipEventElapsedTime(&elapsed, start, stop);
		return elapsed;
	}
};


/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Cuda Kernels

__global__ void Compute_Histogram_512_Cuda(uint32_t* hist, const uint16_t* data, uint16_t width, uint16_t height, uint8_t maxValueBitWidth)
{
	// NOTE: # of bins of histogram must match block size (number of threads in block), and in this case must be 512.
	//		 i.e. the number of threads per block must be the same as the number of bins.

	// maxValueBitWidth = the number of bits needed to represent the max value in the data array.  For example, if the data
	//					  array is built from a 10-bit A-to-D converter, then maxValueBitWidth = 10 since no value will be greather 
	//					  than 2^10.  The minimum value for maxValueBitWidth is driven by the number of bins.  For 256 bins (2^8), 
	//					  the min value is 8.  If bins were 1024 (i.e. 2^10), then the min value for maxValueBitWidth would be 10.

	if (maxValueBitWidth < 8) maxValueBitWidth = 8; // make sure we aren't below the min as described above

	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int nThread = threadIdx.y * blockDim.x + threadIdx.x; // index of thread within block
	int nPixel = y * width + x; // index of pixel within image

	if (x >= width) return;
	if (y >= height) return;

	// if image pixel value == 0, don't add it to the histogram.  Pixel that are 0 are pixels that are outside of the mask 
	// and thus should not be part of the histogram
	if (data[nPixel] == 0) return;

	//Create shared buffer size of threads per block and clear it 
	//Size of array equals numBins 
	__shared__ uint32_t tmpHist[512];
	tmpHist[nThread] = 0;
	__syncthreads();


	//based on the value of this pixel, find the correct bin of the local histogram to increment, and then increment it
	uint8_t shift = maxValueBitWidth - 9;
	int binNumber = data[nPixel] >> shift;

	if (binNumber>511)
	{
		binNumber = 511;
	}

	//float f1 = ((float)(data[nPixel]))/1023.0 * 255;
	//uint8_t binNumber = (uint8_t)f1;


	atomicAdd(&(tmpHist[binNumber]), 1);
	__syncthreads();  // wait for all threads in this block to finish so that the local histogram is finished

	// Update global memory (global histogram)	
	atomicAdd(&(hist[nThread]), tmpHist[nThread]);

}

__global__ void compute_histogram_256_Cuda(uint32_t* hist, const uint16_t* data, uint16_t width, uint16_t height, uint8_t maxValueBitWidth)
{
	// NOTE: # of bins of histogram must match block size (number of threads in block), and in this case must be 256.
	//		 i.e. the number of threads per block must be the same as the number of bins.

	// maxValueBitWidth = the number of bits needed to represent the max value in the data array.  For example, if the data
	//					  array is built from a 10-bit A-to-D converter, then maxValueBitWidth = 10 since no value will be greather 
	//					  than 2^10.  The minimum value for maxValueBitWidth is driven by the number of bins.  For 256 bins (2^8), 
	//					  the min value is 8.  If bins were 1024 (i.e. 2^10), then the min value for maxValueBitWidth would be 10.

	if (maxValueBitWidth < 8) maxValueBitWidth = 8; // make sure we aren't below the min as described above

	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;
	int nThread = threadIdx.y * blockDim.x + threadIdx.x; // index of thread within block
	int nPixel = y * width + x; // index of pixel within image

	if (x >= width) return;
	if (y >= height) return;

	//Create shared buffer size of threads per block and clear it 
	//Size of array equals numBins 
	__shared__ uint32_t tmpHist[256];
	tmpHist[nThread] = 0;
	__syncthreads();


	//based on the value of this pixel, find the correct bin of the local histogram to increment, and then increment it
	uint8_t shift = maxValueBitWidth - 8;
	int binNumber = data[nPixel] >> shift;

	if (binNumber>255)
	{
		binNumber = 255;
	}

	//float f1 = ((float)(data[nPixel]))/1023.0 * 255;
	//uint8_t binNumber = (uint8_t)f1;


	atomicAdd(&(tmpHist[binNumber]), 1);
	__syncthreads();  // wait for all threads in this block to finish so that the local histogram is finished

	// Update global memory (global histogram)	
	atomicAdd(&(hist[nThread]), tmpHist[nThread]);

}

__global__ void MaskImage_Cuda(uint16_t* image, uint16_t* mask, uint16_t width, uint16_t height, float* flatFieldCorrectionArray)
{
	// this function zeroes out all pixels in image that are not in the mask

	// image - a greyscale image with each pixel being a uint16_t
	// mask - a image where pixels with value>0 will be passed through, and pixels with value==0 will be masked out (set to zero).
	//		  The mask is created where pixels with a value of 1, belong in mask aperture 1.  Pixels with value of 2, belong in 
	//		  mask aperture 2...and so on.  
	// width,height - dimensions of image in pixels

	// calc x,y position of pixel to operate on
	uint32_t x = blockIdx.x * blockDim.x + threadIdx.x; // column of pixel inside panel
	uint32_t y = blockIdx.y * blockDim.y + threadIdx.y; // row of pixel inside panel

	// make sure we don't try to operate outside the image
	if (x >= width) return;
	if (y >= height) return;

	// calculate pixel position in array
	uint32_t n = (y * width) + x;

	// apply mask to image
	if (mask[n] == 0)
	{	
		// this pixel is not within a mask aperture, so zero it out
		image[n] = 0;  
	}
	else
	{
		// this pixel is within a mask aperture.  Read the mask aperture value and use that value as an index into the 
		// flat field correction array

		float ffcGain = flatFieldCorrectionArray[mask[n]-1]; // subtract 1 since the mask values are 1-based, and the ffc array is 0-based.
		image[n] = (uint32_t)(ffcGain * ((float)image[n]));
	}
}

__global__ void ConvertGrayscaleToColor_Cuda(uint8_t* color, uint16_t* gray, uint8_t* redMap, uint8_t* greenMap, uint8_t* blueMap,
	uint16_t width, uint16_t height, uint16_t maxGrayValue, uint16_t scaleLower, uint16_t scaleUpper)
{
	// this function converts a grayscale image to a color image using the provided color map

	// color - destination color image (format is ARGB)
	// gray -  source grayscale image
	// redMap, greenMap, blueMap - arrays (maps) that provide color components for each possible grayscale value. For example,
	//							   if a pixel in the gray image has a value = 100, then the corresponding pixel in the color image
	//							   would have its RGB component values set to redMap[100], greenMap[100], and blueMap[100], respectively.
	// width, height - image dimensions
	// maxGrayValue - the maximum possible grayscale value, i.e. length of color map (length of redMap, greenMap, and blueMap)

	// scaleLower, scaleUpper - these values are used to scale the grayscale value of a pixel before it is converted to color.
	//
	//                         scaleUpper
	//						   ________________
	//	maxGrayValue|         /
	//				|        /
	//				|       /
	//				|      /
	//			0	|_____/____________________ 
	//                   scaleLower
	//
	//  Here's the math:
	//		if (pixelValue < scaleLower) set pixelValue = 0
	//      else if (pixelValue < scaleUpper) set pixelValue = maxGrayValue
	//      else 

	// calc x,y position of pixel to operate on
	uint32_t x = blockIdx.x * blockDim.x + threadIdx.x; // column of pixel inside panel
	uint32_t y = blockIdx.y * blockDim.y + threadIdx.y; // row of pixel inside panel

	// make sure we don't try to operate outside the image
	if (x >= width) return;
	if (y >= height) return;

	// calculate pixel position in gray array
	uint32_t nG = (y * width) + x;

	// calculate pixel position in color array
	uint32_t nC = (y * width * 4) + (x * 4);

	// make sure grayscale value is not outside of color maps
	if (gray[nG] > maxGrayValue) gray[nG] = maxGrayValue;

	// scale the value
	uint16_t val = gray[nG];
	if (val < scaleLower) val = 0;
	else if (val >= scaleUpper) val = maxGrayValue;
	else 
		{
			float fval = (float)maxGrayValue/(float)(scaleUpper-scaleLower) * (float)(val-scaleLower);
			val = (uint16_t)fval;
		}

	// set pixel component values for color image
	color[nC + 0] = blueMap[val];	// blue
	color[nC + 1] = greenMap[val];	// green
	color[nC + 2] = redMap[val];	// red
	color[nC + 3] = 255;			// alpha

}

__global__ void CopyCudaArrayToD3D9Memory_Cuda(uint8_t *dest, uint8_t *source, uint16_t pitch, uint16_t width, uint16_t height)
{
	// calc x,y position of pixel to operate on
	uint32_t x = blockIdx.x * blockDim.x + threadIdx.x; // column of pixel inside panel
	uint32_t y = blockIdx.y * blockDim.y + threadIdx.y; // row of pixel inside panel

	// make sure we don't try to operate outside the image
	if (x >= width) return;
	if (y >= height) return;

	// calc position of pixel in cuda array (remember that pitch may not equal width)
	//uint32_t nD = ((height - 1 - y)*pitch) + (x * 4);
	uint32_t nD = (y*pitch) + (x * 4);
	uint32_t nS = (y*width * 4) + (x * 4);

	// copy data
	dest[nD] = source[nS];
	dest[nD + 1] = source[nS + 1];
	dest[nD + 2] = source[nS + 2];
	dest[nD + 3] = source[nS + 3];
}

__global__ void BuildHistogramImage_Cuda(uint8_t* histImage, uint32_t* hist, uint16_t numBins, uint16_t width, uint16_t height, uint32_t maxBinCount)
{
	// this function builds the image for a histogram given by the variable hist.  
	//
	// histImage - the output histogram image.  This is a color image (ARGB, 8 bits per component)
	// hist - is an array which contains the data for the histogram
	// numBins - is the number of bins in the histogram
	// width, height - dimensions of the histImage in pixels
	// maxBinCount - the maximum value that can appear in each bin of the histogram

	// calc x,y position of pixel to operate on
	uint32_t x = blockIdx.x * blockDim.x + threadIdx.x; // column of pixel inside panel
	uint32_t y = blockIdx.y * blockDim.y + threadIdx.y; // row of pixel inside panel

	// make sure we don't try to operate outside the image
	if (x >= width) return;
	if (y >= height) return;

	// calculate the array index into the histogram image
	uint32_t n = (y * width * 4) + (x * 4);  // ARGB image

	// calculate the width of each bin in pixels
	uint16_t binWidth = width / numBins;

	// calculate the bin that this pixel belongs in
	uint16_t binNumber = x / binWidth;
	if (binNumber>numBins) binNumber = numBins;

	// calculate height of the bar for his bin
	uint32_t value = hist[binNumber];  // get the height of the bar for this bin
	uint32_t barHeight = (uint32_t)((float)value * (float)height / (float)maxBinCount);  // calculate the bar height in pixels
	if (barHeight > height) barHeight = height; // make sure the bar height in pixels is not greater than the histogram image height

	// determine if this pixel is in the bar or above it (i.e. determine color of pixel)
	if (y < (height - barHeight)) // pixel is above bar (thus pixel is background color...likely white)
	{   
		histImage[n + 0] = 220;	// blue
		histImage[n + 1] = 220;	// green
		histImage[n + 2] = 220;	// red
		histImage[n + 3] = 255;	// alpha
	}
	else  // pixel is part of bar, so make it the color of the bar (likely black)
	{
		histImage[n + 0] = 0;	// blue
		histImage[n + 1] = 0;	// green
		histImage[n + 2] = 0;	// red
		histImage[n + 3] = 255;	// alpha
	}
}

__global__ void CalcApertureSums_Cuda(uint32_t* sumArray, uint16_t* image, uint16_t* mask, uint16_t width, uint16_t height)
{
	// This function calculate the sum of pixels for each aperture of a mask.  It expects that the mask is formated as follows:
	//		mask pixels with a value of 0 belong to no apertures, thus they will not be part of any sum
	//      mask pixels with a value of 1 belong in aperture 1, which is added to the value in sumArray[0]
	//      mask pixels with a value of 2 belong in aperture 2, which is added to the value in sumArray[1]
	//		and so on...

	// sumArray - output array of the sum of pixel values for each aperature.  For example, for a mask with 24x16 (384) apertures, there
	//			  will be 384 values in sumArray
	// image - input grayscale image from which sums are calculated
	// mask  - input mask that is formatted as described in the description above for this function
	// width, height - dimensions of the image and mask in pixels

	// calc x,y position of pixel to operate on
	uint32_t x = blockIdx.x * blockDim.x + threadIdx.x; // column of pixel inside panel
	uint32_t y = blockIdx.y * blockDim.y + threadIdx.y; // row of pixel inside panel

	// make sure we don't try to operate outside the image
	if (x >= width) return;
	if (y >= height) return;

	// calculate pixel position in image and mask
	uint32_t n = (y * width) + x;

	// get aperture number from mask
	if (mask[n] > 0) // is this pixel inside of any of the apertures of the mask?
	{ // yes
		atomicAdd(&sumArray[mask[n] - 1], image[n]);
	}

	__syncthreads();  // wait for all threads in this block to finish so that the local histogram is finished

}

__global__ void FlatField_Cuda(uint16_t* image, uint16_t* dark, uint16_t* gain, uint16_t width, uint16_t height)
{
	// this function flat field corrects the given grayscale image. It uses the following function:
	//
	//		C[i,j] = ((R[i,j] - D[i,j]) * m) / (F[i,j] - D[i,j]) = (R[i,j] - D[i,j]) * G[i,j]
	//
	//			where G[i,j] = m / (F[i,j] - D[i,j])
	//
	//				  m = average of F-D
	//
	//		i,j = row,column of pixel in image
	//		C = corrected image
	//		R = raw image
	//		F = flat field reference image (evenly illuminated image, meant to show unevenness of illumination)
	//		D = dark field reference image (image taken with no illumination, meant to show distribution of background)
	//		G = gain

	//	parameters passed into function:
	//	image - grayscale image to be corrected.  This is both the input and output image (the input image is over written)
	//  dark  - this is the dark field image (must be same dimensions as image), probably stored in database
	//  gain  - this is the gain array (must be same dimensions as image), that is calculated elsewhere
	//  width, height - dimensions of image (and dark) in pixels

	// calc x,y position of pixel to operate on
	uint32_t x = blockIdx.x * blockDim.x + threadIdx.x; // column of pixel inside panel
	uint32_t y = blockIdx.y * blockDim.y + threadIdx.y; // row of pixel inside panel

	// make sure we don't try to operate outside the image
	if (x >= width) return;
	if (y >= height) return;

	// calculate pixel position in image and dark arrays
	uint32_t n = (y * width) + x;

	image[n] = (image[n] - dark[n]) * gain[n];
}

__global__ void CopyRoiToFullImage_Cuda(uint16_t* full, uint16_t* roi, uint16_t fullW, uint16_t fullH,
	uint16_t  roiX, uint16_t roiY, uint16_t roiW, uint16_t roiH)
{
	// This function is used to copy a ROI image from the camera into a memory space that holds a full frame.
	// It is used when the camera is set up to capture only a part of the CCD (an Region of Interest - ROI), and 
	// since all of the algorithms, kernels, display routines, etc. are set up to handle full frames, this
	// function simply copies the ROI into a full frame.  Pixels outside the ROI are set to zero.

	// calc x,y position of pixel to operate on in the full frame
	uint32_t x = blockIdx.x * blockDim.x + threadIdx.x; // column of pixel inside full frame image
	uint32_t y = blockIdx.y * blockDim.y + threadIdx.y; // row of pixel inside full frame image

	// make sure we don't try to operate outside the full image
	if (x >= fullW) return;
	if (y >= fullH) return;

	// calculate pixel position in arrays
	uint32_t fullN = (y * fullW) + x;  // index into full frame

	// calculate x,y position in ROI
	int32_t xr = x - roiX;
	int32_t yr = y - roiY;

	// are we inside ROI?

	if (x >= roiX && x < (roiX + roiW) && y >= roiY && y < (roiY + roiH))
	{
		uint32_t roiN = (yr * roiW) + xr; // index into roi frame

		// inside ROI
		full[fullN] = roi[roiN];
	}
	else
	{
		// outside ROI
		full[fullN] = 0;
	}
}


/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
/////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// Kernel Calling Functions

void Call_ConvertGrayscaleToColor(uint8_t* color, uint16_t* gray, uint8_t* redMap, uint8_t* greenMap, uint8_t* blueMap,
	uint16_t width, uint16_t height, uint16_t maxGrayValue, uint16_t scaleLower, uint16_t scaleUpper)
{
	dim3 block, grid;
	block.x = 32; block.y = 8; block.z = 1;
	grid.x = width / block.x;
	grid.y = height / block.y;
	grid.z = 1;

	ConvertGrayscaleToColor_Cuda << <grid, block >> >(color, gray, redMap, greenMap, blueMap, width, height, maxGrayValue, scaleLower, scaleUpper);

}

void Call_CopyRoiToFullImage(uint16_t* full, uint16_t* roi, uint16_t fullW, uint16_t fullH,
	uint16_t  roiX, uint16_t roiY, uint16_t roiW, uint16_t roiH)
{
	dim3 block, grid;
	block.x = 32; block.y = 8; block.z = 1;
	grid.x = fullW / block.x;
	grid.y = fullH / block.y;
	grid.z = 1;
	CopyRoiToFullImage_Cuda<<<grid,block>>>(full, roi, fullW, fullH, roiX, roiY, roiW, roiH);
}

void Call_MaskImage(uint16_t* image, uint16_t* mask, uint16_t width, uint16_t height, float* ffcArray)
{
	dim3 block, grid;
	block.x = 32; block.y = 8; block.z = 1;
	grid.x = width / block.x;
	grid.y = height / block.y;
	grid.z = 1;
	MaskImage_Cuda<<<grid,block>>>(image, mask, width, height, ffcArray);
}

void Call_CopyCudaArrayToD3D9Memory(uint8_t* pDest, uint8_t* pSource, uint16_t pitch, uint16_t width, uint16_t height)
{
	hipError_t res = hipDeviceSynchronize();

	dim3 threadsPerBlock(32, 32);  // 32x16 = 512 threads per block	
	dim3 numBlocks((width + threadsPerBlock.x - 1) / threadsPerBlock.x, (height + threadsPerBlock.y - 1) / threadsPerBlock.y);
 	CopyCudaArrayToD3D9Memory_Cuda << <numBlocks, threadsPerBlock >> >(pDest, pSource, pitch, width, height);
}

void Call_ComputeHistogram_512(uint32_t* hist, const uint16_t* data, uint16_t width, uint16_t height, uint8_t maxValueBitWidth)
{
	dim3 block, grid;
	block.x = 32; block.y = 16; block.z = 1; // block size must be 512 = 32 * 16
	grid.x = width / block.x;
	grid.y = height / block.y;
	grid.z = 1;

	Compute_Histogram_512_Cuda<<<grid,block>>>(hist, data, width, height, maxValueBitWidth);
}

void Call_BuildHistogramImage_512(uint8_t* histImage, uint32_t* hist, uint16_t numBins, uint16_t width, uint16_t height, uint32_t maxBinCount)
{
	dim3 block, grid;
	block.x = 32; block.y = 16; block.z = 1;
	grid.x = width / block.x;
	grid.y = height / block.y;
	grid.z = 1;

	BuildHistogramImage_Cuda << <grid, block >> >(histImage, hist, numBins, width, height, maxBinCount);
}

void Call_CalcApertureSums(uint32_t* sumArray, uint16_t* image, uint16_t* mask, uint16_t width, uint16_t height)
{
	dim3 block, grid;
	block.x = 32; block.y = 16; block.z = 1;
	grid.x = width / block.x;
	grid.y = height / block.y;
	grid.z = 1;

	CalcApertureSums_Cuda << <grid, block >> >(sumArray, image, mask, width, height);
}
